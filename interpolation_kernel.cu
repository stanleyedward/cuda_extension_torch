#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename scalar_t> // scalar_t is a placeholder dtype so we dont have to explicitly define the dtype
__global__ void trilinear_forward_kernel(
            const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> features,
            const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
            torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> output
){

}

torch::Tensor trilinear_forward_cu(
    torch::Tensor features,
    torch::Tensor points        
){  
    const int N = features.size(0), F = features.size(2); //  num of cubes and dimension of features in each vertex

    // feat_interp_output = torch.zeros(N, F, dtype=torch.float32, device='cuda:0')
    // torch::zeros({N,F}, torch::dtype(torch::kInt32).device(features.device())); // change tensors dtype and device
    torch::Tensor featInterpOutput = torch::empty({N, F}, features.options()); // options sets dtype and device same as features
    const dim3 numThreadsPerBlock(16, 16, 1); //256 threads in each dim
    const dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, (F + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);

    // instantiate kernel
    AT_DISPATCH_FLOATING_TYPES(features.type(), "trilinear_forward_cu()", 
    ([&] {
        trilinear_forward_kernel<scalar_t><<<numBlocks, numThreadsPerBlock>>>(
            // packed accessor is type conversion for tensors so cuda can manipulate them (not needed by primitive cpp dtypes)
            // restrictPtrTraits: to prevent memory overlay of tensors
            // size_t:  how many steps to take btw each element 
            features.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),         
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            featInterpOutput.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    })  
    );

    return featInterpOutput;
}