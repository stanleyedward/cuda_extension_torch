#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename scalar_t> // scalar_t is a placeholder dtype so we dont have to explicitly define the dtype
__global__ void trilinear_forward_kernel(
            const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> features,
            const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
            torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> output
){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    const int f = blockDim.y * blockIdx.y + threadIdx.y;

    if (n < features.size(0) && f < features.size(2)){
        //since the range for points is [-1, 1] we div by 2 to normalize
        const scalar_t u = (points[f][0] + 1)/2;
        const scalar_t v = (points[f][1] + 1)/2;
        const scalar_t w = (points[f][1] + 1)/2;

        //interpolation coef
        const scalar_t a = (1-v)*(1-w);
        const scalar_t b = (1-v)*w;
        const scalar_t c = v*(1-w);
        const scalar_t d = 1-a-b-c;

        output[n][f] = (1-u)*(a*features[n][0][f] +
                        b*features[n][1][f] +
                        c*features[n][2][f] + 
                        d*features[n][3][f]) +
                        u*(a*features[n][4][f]+
                        b*features[n][5][f]+
                        c*features[n][6][f]+
                        d*features[n][7][f]);

    }
}

torch::Tensor trilinear_forward_cu(
    torch::Tensor features,
    torch::Tensor points        
){  
    const int N = features.size(0), F = features.size(2); //  num of cubes and dimension of features in each vertex

    // feat_interp_output = torch.zeros(N, F, dtype=torch.float32, device='cuda:0')
    // torch::zeros({N,F}, torch::dtype(torch::kInt32).device(features.device())); // change tensors dtype and device
    torch::Tensor featInterpOutput = torch::empty({N, F}, features.options()); // options sets dtype and device same as features
    const dim3 numThreadsPerBlock(16, 16, 1); //256 threads in each dim
    const dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, (F + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);

    // instantiate kernel
    AT_DISPATCH_FLOATING_TYPES(features.type(), "trilinear_forward_cu()", 
    ([&] {
        trilinear_forward_kernel<scalar_t><<<numBlocks, numThreadsPerBlock>>>(
            // packed accessor is type conversion for tensors so cuda can manipulate them (not needed by primitive cpp dtypes)
            // restrictPtrTraits: to prevent memory overlay of tensors
            // size_t:  how many steps to take btw each element 
            features.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),         
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            featInterpOutput.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    })  
    );

    return featInterpOutput;
}